#include "hip/hip_runtime.h"
#include <correspondcuda.h>

using namespace std;

__constant__ lpt::CameraPairCUDA pairs_k[100];

__constant__ int num_matches_k;

__global__ void calcEpipolarResidualAllInOneStreams_kernel(int p, float match_threshold, lpt::KernelArray<float> particles_x, lpt::KernelArray<float> particles_y, lpt::KernelArray<int> num_particles, lpt::KernelArray<lpt::MatchIDs> matches2way, lpt::KernelArray<int> num_matches )
{	
	p += blockIdx.x;
	int b = blockIdx.y * blockDim.x + threadIdx.x;
	int id_b = b;
	if (pairs_k[p].cam_b_id != 0)
		id_b += num_particles.data[pairs_k[p].cam_b_id - 1];
	
	if (id_b < num_particles.data[pairs_k[p].cam_b_id] ) {	
		float line[3];
		float x = particles_x.data[id_b];
		float y = particles_y.data[id_b];
		
		line[0] = pairs_k[p].F[0][0] * x + pairs_k[p].F[1][0] * y + pairs_k[p].F[2][0] * 1.f;
		line[1] = pairs_k[p].F[0][1] * x + pairs_k[p].F[1][1] * y + pairs_k[p].F[2][1] * 1.f;
		line[2] = pairs_k[p].F[0][2] * x + pairs_k[p].F[1][2] * y + pairs_k[p].F[2][2] * 1.f;
		float factor = line[0] * line[0] + line[1] * line[1];
		factor = factor ? 1.f/sqrtf(factor) : 1.f;
		line[0] *= factor;
		line[1] *= factor;
		line[2] *= factor;
     
		int id_a_start = 0;
		if (pairs_k[p].cam_a_id != 0) 
			id_a_start = num_particles.data[ pairs_k[p].cam_a_id - 1];
		int id_a_end = num_particles.data[ pairs_k[p].cam_a_id ];
		
		int match_id = b;
		if (p !=0 )
			match_id += num_matches.data[p - 1];
		int match_count = 0; 
		for (int id_a = id_a_start; id_a < id_a_end; ++id_a) {		
			bool matched = static_cast<int>( floor( match_threshold / fabs( particles_x.data[id_a] * line[0]  + particles_y.data[id_a] * line[1] + line[2] ) ) );
			if ( matched && match_count < num_matches_k ) { 
				matches2way.data[match_id].ids[match_count] = id_a;
				++match_count;
			}
		}
	}
}

__global__ void calcEpipolarResidualAllInOne_kernel( float match_threshold, lpt::KernelArray<float> particles_x, lpt::KernelArray<float> particles_y, lpt::KernelArray<int> num_particles, lpt::KernelArray<lpt::MatchIDs> matches2way, lpt::KernelArray<int> num_matches )
{	
	int p = blockIdx.x; 
	int b = blockIdx.y * blockDim.x + threadIdx.x;
	int id_b = b;
	if (pairs_k[p].cam_b_id != 0)
		id_b = num_particles.data[pairs_k[p].cam_b_id - 1] + b;
	
	if (id_b < num_particles.data[pairs_k[p].cam_b_id] ) {	
		float line[3];
		float x = particles_x.data[id_b]; 
		float y = particles_y.data[id_b]; 
		
		line[0] = pairs_k[p].F[0][0] * x + pairs_k[p].F[1][0] * y + pairs_k[p].F[2][0] * 1.f;
		line[1] = pairs_k[p].F[0][1] * x + pairs_k[p].F[1][1] * y + pairs_k[p].F[2][1] * 1.f;
		line[2] = pairs_k[p].F[0][2] * x + pairs_k[p].F[1][2] * y + pairs_k[p].F[2][2] * 1.f;
		float factor = line[0] * line[0] + line[1] * line[1];
		factor = factor ? 1.f/sqrtf(factor) : 1.f;
		line[0] *= factor;
		line[1] *= factor;
		line[2] *= factor;
     
		int id_a_start = 0;
		if (pairs_k[p].cam_a_id != 0) 
			id_a_start = num_particles.data[ pairs_k[p].cam_a_id - 1];
		int id_a_end = num_particles.data[ pairs_k[p].cam_a_id ];
		int match_id = b;
		if (p !=0 )
			match_id += num_matches.data[p - 1];
		int match_count = 0; 
		for (int id_a = id_a_start; id_a < id_a_end; ++id_a) {		
			bool matched = static_cast<int>( floor( match_threshold / fabs( particles_x.data[id_a] * line[0]  + particles_y.data[id_a] * line[1] + line[2] ) ) );
			if ( matched && match_count < num_matches_k ) { 
				matches2way.data[match_id].ids[match_count] = id_a;
				++match_count;
			}
		}
	}
}

__global__ void calcEpipolarLines_kernel(lpt::KernelArray<float> particles_x, lpt::KernelArray<float> particles_y, lpt::KernelArray<int> num_particles, lpt::KernelArray<float> lines_x, lpt::KernelArray<float> lines_y, lpt::KernelArray<float> lines_z, lpt::KernelArray<int> num_lines )
{	
	int p = blockIdx.x; 
	int b = blockIdx.y * blockDim.x + threadIdx.x;
	int cam_b_id = pairs_k[p].cam_b_id;
	int id_b = b;
	if (cam_b_id != 0)
		id_b = num_particles.data[cam_b_id - 1] + b;

	if (id_b < num_particles.data[cam_b_id] ) {

		float x = particles_x.data[id_b];
		float y = particles_y.data[id_b];
		int line_id = b;
		if (p !=0 )
			line_id += num_lines.data[p -1];

		lines_x.data[line_id] = pairs_k[p].F[0][0] * x + pairs_k[p].F[1][0] * y + pairs_k[p].F[2][0] * 1.f;
		lines_y.data[line_id] = pairs_k[p].F[0][1] * x + pairs_k[p].F[1][1] * y + pairs_k[p].F[2][1] * 1.f;
		lines_z.data[line_id] = pairs_k[p].F[0][2] * x + pairs_k[p].F[1][2] * y + pairs_k[p].F[2][2] * 1.f;
		float factor = lines_x.data[line_id] * lines_x.data[line_id] + lines_y.data[line_id] * lines_y.data[line_id];
		factor = factor ? 1.f/sqrtf(factor) : 1.f;
		lines_x.data[line_id] *= factor;
		lines_y.data[line_id] *= factor;
		lines_z.data[line_id] *= factor; 
	}
}

__global__ void calcEpipolarResiduals_kernel(float match_threshold, lpt::KernelArray<float> particles_x, lpt::KernelArray<float> particles_y, lpt::KernelArray<int> num_particles, lpt::KernelArray<float> lines_x, lpt::KernelArray<float> lines_y, lpt::KernelArray<float> lines_z, lpt::KernelArray<int> num_lines, lpt::KernelArray<lpt::MatchIDs> matches2way, lpt::KernelArray<int> num_matches)
{
	////__shared__ float line[3];
	//
	//int line_id = blockIdx.x;
	////float line[] = {lines_x.data[line_id], lines_y.data[line_id], lines_z.data[line_id]};
	//int cam_id;
	//int r_id = residuals.size;
	//for (int pair_id = 0; pair_id < num_lines.size; ++pair_id) {
	//	if( line_id < num_lines.data[pair_id] ) {
	//		cam_id = pairs_k[pair_id].cam_a_id;
	//		if (pair_id !=0 )
	//			r_id = num_residuals.data[pair_id - 1] + (line_id - num_lines.data[pair_id -1]) * (num_particles.data[cam_id] - num_particles.data[cam_id - 1]) + threadIdx.x;
	//		else
	//			r_id = (line_id - num_lines.data[pair_id -1]) * (num_particles.data[cam_id] - num_particles.data[cam_id - 1]) + threadIdx.x;
	//		break;
	//	}
	//}

	////if (threadIdx.x == 0) {
	////	line[0] = lines_x.data[line_id];
	////	line[1] = lines_y.data[line_id];
	////	line[2] = lines_z.data[line_id];
	////}	
	////__syncthreads(); //FIXME: put lines in constant memory if possible

	//int id_a = num_particles.data[cam_id - 1] + blockIdx.y * blockDim.x + threadIdx.x;
	//
	//if ( id_a < num_particles.data[cam_id] && r_id < residuals.size ) {
	//	residuals.data[r_id] = static_cast<int>( match_threshold / fabs( particles_x.data[id_a] * lines_x.data[line_id]  + particles_y.data[id_a] * lines_y.data[line_id] + lines_z.data[line_id] ) );
	//}

}

namespace lpt {

PointMatcherCUDA::PointMatcherCUDA() {
	cout << "Epipolor Point matcher created (CUDA Enabled)" << endl;
	int devcount = 0;
	hipGetDeviceCount(&devcount);
	for (int i = 0; i < devcount; ++i) {
		hipDeviceProp_t device_prop;
		hipGetDeviceProperties(&device_prop, i);
		if (! device_prop.kernelExecTimeoutEnabled ) {
			cout << "Device " << i << ":  " <<  device_prop.name << "  added to available queue" <<endl;
		} else {
			cout << "Device " << i << ":  " << device_prop.name << " added to available queue (Kernel run time limited)" << endl;
		}
		compute_devices_available.push(i);
	}
}

int PointMatcherCUDA::getNextComputeDeviceID() {
	boost::mutex::scoped_lock(this->mutex);
	int id = this->compute_devices_available.front();
	this->compute_devices_available.pop();
	return id;
}

void PointMatcherCUDA::initializeEpipolarMatchThread(int thread_id) {
	auto& cameras = shared_objects->cameras;
	auto& camera_pairs = shared_objects->camera_pairs;
	int id = getNextComputeDeviceID();
	hipSetDevice( id );
	cout << "PointMatcherCUDA Thread " << thread_id << " setting device " << id << endl;

	particles_x_h.resize(this->initial_max_particles_per_image * cameras.size(), 0.f );
	particles_x_d = particles_x_h;
	
	particles_y_h.resize(this->initial_max_particles_per_image * cameras.size(), 0.f );
	particles_y_d = particles_y_h;
	
	num_particles_h.resize(cameras.size(), 0 );
	num_particles_d = num_particles_h;

	camera_pairs_h.resize( camera_pairs.size() );
	
	num_matches_h.resize( camera_pairs.size(), 0 );
	num_matches_d = num_matches_h;

	for (int i = 0; i < NUM_MATCHES; ++i)
		this->match_initializer.ids[i] = -1;

	matches2way_h.resize( camera_pairs.size() * this->initial_max_particles_per_image, this->match_initializer);
	matches2way_d = matches2way_h;

	for (int i = 0; i < camera_pairs.size(); ++i) {
		for (int n = 0; n < 3; ++n)
			for (int m = 0; m < 3; ++m)
				camera_pairs_h[i].F[n][m] = static_cast<float>(camera_pairs[i].F[n][m]);
		camera_pairs_h[i].cam_a_id = camera_pairs[i].cam_A.id;
		camera_pairs_h[i].cam_b_id = camera_pairs[i].cam_B.id;
	}
			
	streams.clear();
	for (int f = 2; f < camera_pairs_h.size(); ++f) {
		//if ( camera_pairs_h.size() % f == 0 ) {
			streams.resize(f);
			//break;
		//}
	}
	cout << "Streams size = " << streams.size() << endl;
	for(int i = 0; i < streams.size(); ++i) 
        hipStreamCreate(&(streams[i]));

	int num = NUM_MATCHES;

	hipMemcpyToSymbol(HIP_SYMBOL( "num_matches_k"), &num,  sizeof(int) );
	hipMemcpyToSymbol(HIP_SYMBOL( "pairs_k"), thrust::raw_pointer_cast(&camera_pairs_h[0]),  sizeof(CameraPairCUDA) * camera_pairs_h.size() );
}

void PointMatcherCUDA::initialize() {
	this->initializeMatchMap();
}

void PointMatcherCUDA::addControls() {
	void* matcher_void_ptr = static_cast<void*> ( this );
	cv::createTrackbar("Match Thresh", string() , &params.match_thresh_level, 100, callbackMatchThreshcuda, matcher_void_ptr);
}

void PointMatcherCUDA::findEpipolarMatches(const lpt::ImageFrameGroup& frame_group, lpt::MatchMap& matchmap) {
	
	thrust::fill(matches2way_d.begin(), matches2way_d.end(), match_initializer);
	
	num_particles_h[0] = frame_group[0].particles.size();
	for(int p = 0; p < frame_group[0].particles.size(); ++p) {
			particles_x_h[p] = static_cast<float>(frame_group[0].particles[p]->x);
			particles_y_h[p] = static_cast<float>(frame_group[0].particles[p]->y);
	}

	int max_particles = num_particles_h[0];
	for(int i = 1; i < frame_group.size(); ++i) {
		num_particles_h[i] = frame_group[i].particles.size() + num_particles_h[i-1];
		for(int p = 0; p < frame_group[i].particles.size(); ++p) {
			particles_x_h[ num_particles_h[i-1] + p] = static_cast<float>(frame_group[i].particles[p]->x);
			particles_y_h[ num_particles_h[i-1] + p] = static_cast<float>(frame_group[i].particles[p]->y);
		}
		if (frame_group[i].particles.size() > max_particles) {
			max_particles = frame_group[i].particles.size();
			if ( max_particles > this->initial_max_particles_per_image) 
				cout << "WARNING IMAGE FRAME HAS EXCEEDED MAX PARTICLES:  correspondcuda.cu" << endl;
		}
	}

	hipMemcpyAsync(thrust::raw_pointer_cast(&num_particles_d[0]), thrust::raw_pointer_cast(&num_particles_h[0]), num_particles_h.size() * sizeof(int), hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(thrust::raw_pointer_cast(&particles_x_d[0]), thrust::raw_pointer_cast(&particles_x_h[0]), *num_particles_h.rbegin() * sizeof(float), hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(thrust::raw_pointer_cast(&particles_y_d[0]), thrust::raw_pointer_cast(&particles_y_h[0]), *num_particles_h.rbegin() * sizeof(float), hipMemcpyHostToDevice, streams[0]);

	num_matches_h[0] = frame_group[camera_pairs_h[0].cam_b_id].particles.size(); 
	for(int i = 1; i < this->camera_pairs_h.size(); ++i ) {
		num_matches_h[i] = frame_group[camera_pairs_h[i].cam_b_id].particles.size() + num_matches_h[i-1]; 
	}

	hipMemcpyAsync(thrust::raw_pointer_cast(&num_matches_d[0]), thrust::raw_pointer_cast(&num_matches_h[0]), num_matches_h.size() * sizeof(int), hipMemcpyHostToDevice, streams[0]);
	
	int num_pairs = camera_pairs_h.size();

	dim3 dimblock(128,1,1);
	dim3 dimgrid( static_cast<unsigned int>(num_pairs), (static_cast<unsigned int>(max_particles) / dimblock.x ) + 1 );
	
	calcEpipolarResidualAllInOne_kernel <<< dimgrid, dimblock, 0, streams[0] >>> (params.match_threshold, particles_x_d, particles_y_d, num_particles_d, matches2way_d, num_matches_d);
	
	hipMemcpyAsync(thrust::raw_pointer_cast(&matches2way_h[0]), thrust::raw_pointer_cast(&matches2way_d[0]), *num_matches_h.rbegin() * sizeof(MatchIDs), hipMemcpyDeviceToHost, streams[0]);
	
	hipStreamSynchronize(streams[0]);
	
	int match_overload = 0;
	for (int p = 0; p < camera_pairs_h.size(); ++p) {
		int match_id = (p == 0 ? 0 : num_matches_h[p-1]);
		int cam_b = camera_pairs_h[p].cam_b_id; 
		int cam_a = camera_pairs_h[p].cam_a_id;
		int b_end = num_particles_h[cam_b];
		int b_start = (cam_b !=0 ? num_particles_h[cam_b-1] : 0);
		int a_start = (cam_a !=0 ? num_particles_h[cam_a-1] : 0);
		for (int b_id = b_start; b_id < b_end; ++b_id, ++match_id) {
			for (int m = 0; m < NUM_MATCHES; ++m) {
				int a_id = matches2way_h[match_id].ids[m];
				if (a_id >= 0) {
					//matchmap[b_id][cam_a][m] = a_id - a_start;
					//matchmap[a_id][cam_b][m] = b_id - b_start; 
					auto itb = std::find(matchmap[b_id][cam_a].begin(), matchmap[b_id][cam_a].end(), -1);
					auto ita = std::find(matchmap[a_id][cam_b].begin(), matchmap[a_id][cam_b].end(), -1);
					if (itb != matchmap[b_id][cam_a].end() && ita != matchmap[a_id][cam_b].end() ) {
						*itb = a_id - static_cast<int>(a_start);
						*ita = static_cast<int>(b_id - b_start);
					} else
						match_overload++;
				}
				else				
					break;
			}				
		}
	}
	if (match_overload > 0)
		;//cout << "WARNING: MORE MATCHES THAN ARRAY SIZE NUM_MATCHES: total overload = " << match_overload << endl;
}

void PointMatcherCUDA::findUniqueMatches(const lpt::ImageFrameGroup& frame_group, lpt::MatchMap& matchmap, vector<lpt::Match::Ptr>& matches) {
	vector<int> num_particles(frame_group.size());
	num_particles[0] = frame_group[0].particles.size();
	for(int i = 1; i < frame_group.size(); ++i) 
		num_particles[i] = frame_group[i].particles.size() + num_particles[i-1];

    for (int cam_a = 0; cam_a < frame_group.size() - 3; ++cam_a) {
		int a_start = (cam_a !=0 ? num_particles[cam_a - 1] : 0);
		for (int a = 0; a < frame_group[cam_a].particles.size(); ++a) {
            lpt::ParticleImage::Ptr Pa = frame_group[cam_a].particles[a];
			if( ! Pa->is_4way_matched ) 
            for (int cam_b = cam_a + 1; cam_b < frame_group.size() - 2; ++cam_b) {
				int b_start = (cam_b !=0 ? num_particles[cam_b-1] : 0);
                for(int match_ab = 0; match_ab < NUM_MATCHES; ++match_ab) { //loop through all A,B matches
					int b = matchmap[a + a_start][cam_b][match_ab]; 
					if (b < 0)
						break;
					lpt::ParticleImage::Ptr Pb = frame_group[cam_b].particles[b];
						
					if( ! Pb->is_4way_matched ) 
					for (int cam_c = cam_b + 1; cam_c < frame_group.size() - 1; ++cam_c) {
                        int c_start = (cam_c !=0 ? num_particles[cam_c-1] : 0);
						for (int match_bc = 0; match_bc < NUM_MATCHES; ++match_bc) {
                            int c = matchmap[b + b_start][cam_c][match_bc];
							if (c < 0) 
								break;
								
							lpt::ParticleImage::Ptr Pc = frame_group[cam_c].particles[c];

							if( ! Pc->is_4way_matched && std::count(matchmap[a + a_start][cam_c].begin(), matchmap[a + a_start][cam_c].end(), c) )  
                            for (int cam_d = cam_c + 1; cam_d < frame_group.size(); ++cam_d) {
								vector<lpt::Match::Ptr> matches4way;
                                int d_start = (cam_d !=0 ? num_particles[cam_d-1] : 0);
								for (int match_cd = 0; match_cd < NUM_MATCHES; ++match_cd) {
									int d = matchmap[c + c_start][cam_d][match_cd];
									if (d < 0)
										break;
									lpt::ParticleImage::Ptr Pd = frame_group[cam_d].particles[d];
									if( ! Pd->is_4way_matched && std::count(matchmap[a + a_start][cam_d].begin(), matchmap[a + a_start][cam_d].end(), d)  && std::count(matchmap[b + b_start][cam_d].begin(), matchmap[b+b_start][cam_d].end(), d)  ) {
										if(! Pa->is_4way_matched && ! Pb->is_4way_matched && ! Pc->is_4way_matched && ! Pd->is_4way_matched) { 
											lpt::Match::Ptr newmatch = lpt::Match::create();
											newmatch->addParticle(Pa,cam_a);
											newmatch->addParticle(Pb,cam_b);
											newmatch->addParticle(Pc,cam_c);
											newmatch->addParticle(Pd,cam_d);
											matches4way.push_back(std::move(newmatch));
											Pa->is_4way_matched = true;
											Pb->is_4way_matched = true;
											Pc->is_4way_matched = true;
											Pd->is_4way_matched = true;
											match_ab = NUM_MATCHES;
											match_bc = NUM_MATCHES;
											match_cd = NUM_MATCHES;
												
										}
                                    } 
                                }
								std::move(matches4way.begin(), matches4way.end(), std::back_inserter(matches) );
                            }
                        }
                    }
                }
            }
		}
    }
}

void PointMatcherCUDA::find3WayMatches(const lpt::ImageFrameGroup& frame_group, lpt::MatchMap& matchmap, vector<lpt::Match::Ptr>& matches) {
	vector<int> num_particles(frame_group.size());
	num_particles[0] = frame_group[0].particles.size();
	for(int i = 1; i < frame_group.size(); ++i) 
		num_particles[i] = frame_group[i].particles.size() + num_particles[i-1];
	
	int num_cameras = frame_group.size();
	matches.clear();

	for (int cam_a = 0; cam_a < frame_group.size() - 2; ++cam_a) {
		int a_start = (cam_a !=0 ? num_particles[cam_a - 1] : 0);
		for (int a = 0; a < frame_group[cam_a].particles.size(); ++a) {
            lpt::ParticleImage::Ptr Pa = frame_group[cam_a].particles[a];
			if( ! Pa->is_4way_matched ) {
                for (int cam_b = cam_a + 1; cam_b < frame_group.size() - 1; ++cam_b) {
					int b_start = (cam_b !=0 ? num_particles[cam_b-1] : 0);
                    for(int match_ab = 0; match_ab < NUM_MATCHES; ++match_ab) { //loop through all A,B matches
						int b = matchmap[a + a_start][cam_b][match_ab]; 
						if (b < 0)
							break;
						lpt::ParticleImage::Ptr Pb = frame_group[cam_b].particles[b];
						
						if( ! Pb->is_4way_matched ) {
							for (int cam_c = cam_b + 1; cam_c < frame_group.size(); ++cam_c) {
								int c_start = (cam_c !=0 ? num_particles[cam_c-1] : 0);
								for (int match_bc = 0; match_bc < NUM_MATCHES; ++match_bc) {
									int c = matchmap[b + b_start][cam_c][match_bc];
									if (c < 0) 
										break;
								
									lpt::ParticleImage::Ptr Pc = frame_group[cam_c].particles[c];

									if( ! Pc->is_4way_matched && std::count(matchmap[a + a_start][cam_c].begin(), matchmap[a + a_start][cam_c].end(), c) ) {
										lpt::Match::Ptr newmatch = lpt::Match::create();
										newmatch->addParticle(Pa,cam_a);
										newmatch->addParticle(Pb,cam_b);
										newmatch->addParticle(Pc,cam_c);

										matches.push_back(std::move(newmatch));
												
										Pa->is_4way_matched = true;
										Pb->is_4way_matched = true;
										Pc->is_4way_matched = true;

										match_ab = NUM_MATCHES;
										match_bc = NUM_MATCHES;
												
										cam_b = num_cameras;
										cam_c = num_cameras;
									}
                                }
                            }
                        }
                    }
                }
			}
		}
	}
	//cout << matches.size() << endl;
}

void PointMatcherCUDA::findEpipolarMatchesStreams(lpt::ImageFrameGroup& frame_group, lpt::MatchMap& matchmap) {
	thrust::fill(matches2way_d.begin(), matches2way_d.end(), match_initializer);
	num_particles_h[0] = frame_group[0].particles.size();
	for(int p = 0; p < frame_group[0].particles.size(); ++p) {
			particles_x_h[p] = static_cast<float>(frame_group[0].particles[p]->x);
			particles_y_h[p] = static_cast<float>(frame_group[0].particles[p]->y);
	}

	int max_particles = num_particles_h[0];
	for(int i = 1; i < frame_group.size(); ++i) {
		num_particles_h[i] = frame_group[i].particles.size() + num_particles_h[i-1];
		for(int p = 0; p < frame_group[i].particles.size(); ++p) {
			particles_x_h[ num_particles_h[i-1] + p] = static_cast<float>(frame_group[i].particles[p]->x);
			particles_y_h[ num_particles_h[i-1] + p] = static_cast<float>(frame_group[i].particles[p]->y);
		}
		if (frame_group[i].particles.size() > max_particles)
			max_particles = frame_group[i].particles.size();
	}
	
	hipMemcpyAsync(thrust::raw_pointer_cast(&num_particles_d[0]), thrust::raw_pointer_cast(&num_particles_h[0]), num_particles_h.size() * sizeof(int), hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(thrust::raw_pointer_cast(&particles_x_d[0]), thrust::raw_pointer_cast(&particles_x_h[0]), *num_particles_h.rbegin() * sizeof(float), hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(thrust::raw_pointer_cast(&particles_y_d[0]), thrust::raw_pointer_cast(&particles_y_h[0]), *num_particles_h.rbegin() * sizeof(float), hipMemcpyHostToDevice, streams[0]);

	int num_pairs = camera_pairs_h.size();
	dim3 dimblock(128,1,1);
	dim3 dimgrid( static_cast<unsigned int>(num_pairs / streams.size()), (static_cast<unsigned int>(max_particles) / dimblock.x ) + 1, 1 );
	 
	num_matches_h[0] = frame_group[camera_pairs_h[0].cam_b_id].particles.size(); 
	for(int i = 1; i < this->camera_pairs_h.size(); ++i ) {
		num_matches_h[i] = frame_group[camera_pairs_h[i].cam_b_id].particles.size() + num_matches_h[i-1]; 
	}

	hipMemcpyAsync(thrust::raw_pointer_cast(&num_matches_d[0]), thrust::raw_pointer_cast(&num_matches_h[0]), num_matches_h.size() * sizeof(int), hipMemcpyHostToDevice, streams[0]);
	
    for(int i = 0; i < streams.size(); i++) 
		calcEpipolarResidualAllInOneStreams_kernel <<< dimgrid, dimblock, 0, streams[i] >>> (i * dimgrid.x , params.match_threshold, particles_x_d, particles_y_d, num_particles_d, matches2way_d, num_matches_d);

	for(int i = 0; i < streams.size(); i++) {
		 int index = (i == 0 ? 0 : num_matches_h[ i * dimgrid.x - 1]);
		 int nbytes = ( i == 0 ?  num_matches_h[dimgrid.x - 1] :  num_matches_h[(i+1)*dimgrid.x - 1] -  num_matches_h[i*dimgrid.x - 1]) * sizeof(MatchIDs);
		 hipMemcpyAsync(thrust::raw_pointer_cast(&matches2way_h[index]), thrust::raw_pointer_cast(&matches2way_d[index]), nbytes, hipMemcpyDeviceToHost, streams[i]);
	}
	
	int match_overload = 0;
	for (unsigned int i = 0; i < streams.size(); ++i) {
		hipStreamSynchronize(streams[i]);
		for (unsigned int p = i * dimgrid.x; p < (i + 1) * dimgrid.x; ++p) {
			int match_id = (p == 0 ? 0 : num_matches_h[p-1]);
			int cam_b = camera_pairs_h[p].cam_b_id; 
			int cam_a = camera_pairs_h[p].cam_a_id;
			int b_end = num_particles_h[cam_b];
			int b_start = (cam_b !=0 ? num_particles_h[cam_b-1] : 0);
			int a_start = (cam_a !=0 ? num_particles_h[cam_a-1] : 0);
			for (int b_id = b_start; b_id < b_end; ++b_id, ++match_id) {
				for (int m = 0; m < NUM_MATCHES; ++m) {
					int a_id = matches2way_h[match_id].ids[m];
					if (a_id >= 0) {
						//matchmap[b_id][cam_a][m] = a_id - a_start;
						//matchmap[a_id][cam_b][m] = b_id - b_start; 
						auto itb = std::find(matchmap[b_id][cam_a].begin(), matchmap[b_id][cam_a].end(), -1);
						auto ita = std::find(matchmap[a_id][cam_b].begin(), matchmap[a_id][cam_b].end(), -1);
						if (itb != matchmap[b_id][cam_a].end() && ita != matchmap[a_id][cam_b].end() ) {
							*itb = a_id - static_cast<int>(a_start);
							*ita = static_cast<int>(b_id - b_start);
						} else
							match_overload++;
					}
					else				
						break;
				}				
			}
		}
	}

	if (match_overload > 0)
		;//cout << "WARNING: MORE MATCHES THAN ARRAY SIZE NUM_MATCHES: total overload = " << match_overload << endl;
}

void PointMatcherCUDA::findEpipolarMatchesManyThreads(lpt::ImageFrameGroup& frame_group) {
	
	num_particles_h[0] = frame_group[0].particles.size();
	for(int p = 0; p < frame_group[0].particles.size(); ++p) {
			particles_x_h[p] = static_cast<float>(frame_group[0].particles[p]->x);
			particles_y_h[p] = static_cast<float>(frame_group[0].particles[p]->y);
	}

	int max_particles = num_particles_h[0];
	for(int i = 1; i < frame_group.size(); ++i) {
		num_particles_h[i] = frame_group[i].particles.size() + num_particles_h[i-1];
		for(int p = 0; p < frame_group[i].particles.size(); ++p) {
			particles_x_h[ num_particles_h[i-1] + p] = static_cast<float>(frame_group[i].particles[p]->x);
			particles_y_h[ num_particles_h[i-1] + p] = static_cast<float>(frame_group[i].particles[p]->y);
		}
		if (frame_group[i].particles.size() > max_particles)
			max_particles = frame_group[i].particles.size();
	}
		
	hipMemcpyAsync(thrust::raw_pointer_cast(&num_particles_d[0]), thrust::raw_pointer_cast(&num_particles_h[0]), num_particles_h.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(thrust::raw_pointer_cast(&particles_x_d[0]), thrust::raw_pointer_cast(&particles_x_h[0]), *num_particles_h.rbegin() * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(thrust::raw_pointer_cast(&particles_y_d[0]), thrust::raw_pointer_cast(&particles_y_h[0]), *num_particles_h.rbegin() * sizeof(float), hipMemcpyHostToDevice);

	int num_pairs = camera_pairs_h.size();
	dim3 dimblock(256);
	dim3 dimgrid( static_cast<unsigned int>(num_pairs), (static_cast<unsigned int>(max_particles) / dimblock.x ) + 1 );

	thrust::host_vector<int> num_lines_h(camera_pairs_h.size(), 0);
	num_lines_h[0] = frame_group[camera_pairs_h[0].cam_b_id].particles.size();
	for(int i = 1; i < this->camera_pairs_h.size(); ++i ) 
		num_lines_h[i] = frame_group[camera_pairs_h[i].cam_b_id].particles.size() + num_lines_h[i-1];
	
	thrust::device_vector<int> num_lines_d = num_lines_h;
	thrust::device_vector<float> lines_x( *num_lines_h.rbegin(), 0.f );
	thrust::device_vector<float> lines_y( *num_lines_h.rbegin(), 0.f );
	thrust::device_vector<float> lines_z( *num_lines_h.rbegin(), 0.f );
	
//	lines_x.resize( *num_lines_h.rbegin(), 0.f );
//	lines_y.resize( *num_lines_h.rbegin(), 0.f );
//	lines_z.resize( *num_lines_h.rbegin(), 0.f );	
	
	calcEpipolarLines_kernel <<< dimgrid, dimblock >>> (particles_x_d, particles_y_d, num_particles_d, lines_x, lines_y, lines_z, num_lines_d);
	
	num_matches_h[0] = frame_group[camera_pairs_h[0].cam_b_id].particles.size(); 
	for(int i = 1; i < this->camera_pairs_h.size(); ++i ) {
		num_matches_h[i] = frame_group[camera_pairs_h[i].cam_b_id].particles.size() + num_matches_h[i-1]; 
	}

	hipMemcpyAsync(thrust::raw_pointer_cast(&num_matches_d[0]), thrust::raw_pointer_cast(&num_matches_h[0]), num_matches_h.size() * sizeof(int), hipMemcpyHostToDevice);
	hipStreamSynchronize(0);
	
	dim3 dimblock2(512,1,1);
	dim3 dimgrid2( static_cast<unsigned int>(*num_lines_h.rbegin()), ( static_cast<unsigned int>(max_particles) / dimblock2.x ) + 1, 1 );
	//cout <<"K2 Grid = " << dimgrid2.x << " x " << dimgrid2.y << " x " << dimgrid2.z << endl;
	//cout <<"K2 Block = " << dimblock2.x << " x " << dimblock2.y << " x " << dimblock2.z << endl;
	
	calcEpipolarResiduals_kernel <<< dimgrid2, dimblock2 >>> (params.match_threshold, particles_x_d, particles_y_d, num_particles_d, lines_x, lines_y, lines_z, num_lines_d, matches2way_d, num_matches_d);

	thrust::copy(matches2way_d.begin(), matches2way_d.begin() + *num_matches_h.rbegin(), matches2way_h.begin() );

}

}
